#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>

#include "dmat.h"

#define NUM_THREADS 32

// Space for the vector data
__constant__ float * distance_vg_a_d;

// Space for the resulting distance
__device__ float * distance_d_d;

void checkCudaError(const char * msg) {
	hipError_t err = hipGetLastError();
	if(hipSuccess != err) {
		if(msg != NULL) {
			printf("[WARNING] %s\n", msg);
		}
		printf("[ERROR] %s : %s\n", msg, hipGetErrorString(err));
		exit(1);
	}
}

__global__ void euclidean_kernel_same(	const float * vg_a, size_t pitch_a, size_t n_a, size_t k,
										float * d, size_t pitch_d )
{
	size_t x = blockIdx.x, y = blockIdx.y;

	if((x == y) && (x < n_a) && (threadIdx.x == 0))
		d[y * pitch_d + x] = 0.0;
  
	// If all element is to be computed
	if(y < n_a && x < y) {
		__shared__ float temp[NUM_THREADS];    

		temp[threadIdx.x] = 0.0;
    
		for(size_t offset = threadIdx.x; offset < k; offset += NUM_THREADS) {
			float t = abs(vg_a[x * pitch_a + offset] - vg_a[y * pitch_a + offset]);
			temp[threadIdx.x] += (t * t);
		}
    
		// Sync with other threads
		__syncthreads();
    
		// Reduce
		for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
			if(threadIdx.x < stride)
				temp[threadIdx.x] += temp[threadIdx.x + stride];
			__syncthreads();
		}
	    
		// Write to global memory
		if(threadIdx.x == 0) {
			float s = sqrt(temp[0]);
			d[y * pitch_d + x] = s;
			d[x * pitch_d + y] = s;
		}
	}
}

void distance_device(	const float * vg_a_d, size_t pitch_a, size_t n_a, size_t k,
						float * d_d, size_t pitch_d) {

	dim3 block(NUM_THREADS, 1, 1);
	dim3 grid(n_a, n_a, 1);

	size_t fbytes = sizeof(float);

	pitch_a /= fbytes;
	pitch_d /= fbytes;

	euclidean_kernel_same<<<grid, block>>>(	vg_a_d, pitch_a, n_a, k,
											d_d, pitch_d);
}

void distanceGPU(	const float *vg_a, size_t pitch_a, size_t n_a, size_t k, 
				float * d, size_t pitch_d ) {
	
	size_t pitch_a_d, pitch_d_d;
	
	// Allocate space for the vectors and distances on the gpu
	hipMallocPitch((void**)&distance_vg_a_d, &pitch_a_d, k * sizeof(float), n_a);
	hipMemcpy2D(distance_vg_a_d, pitch_a_d, vg_a, pitch_a, k * sizeof(float), n_a, hipMemcpyHostToDevice);

	hipMallocPitch((void**)&distance_d_d, &pitch_d_d, n_a * sizeof(float), n_a);
	
	checkCudaError("distance function : malloc and memcpy");
    
	distance_device(distance_vg_a_d, pitch_a_d, n_a, k,
					distance_d_d, pitch_d_d);
	
	checkCudaError("distance function : kernel invocation");

	// Copy the result back to cpu land now that gpu work is done
	hipMemcpy2D(d, pitch_d, distance_d_d, pitch_d_d, n_a * sizeof(float), n_a, hipMemcpyDeviceToHost);
	checkCudaError("distance function : memcpy");
    
	// Free allocated space
	hipFree(distance_vg_a_d);
	hipFree(distance_d_d);
}